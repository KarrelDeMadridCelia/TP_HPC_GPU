
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>

#define THREAD_PER_BLOCK 16 // on fixe le nombre de colonnes à 16
#define COLUMNS 16



//fct gpu

__global__ void multiplication_matrix_GPU(int *a, int *b, int*c)
{
	int idx = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

	int sum = 0;

        __shared__ int bs[16]; // définition de la mémoire partagée
 
        bs[threadIdx.x]= b[threadIdx.x];


	//for(int j = 0; j<COLUMNS;++j,++a,++b)
	//	sum += (a[idx*COLUMNS+j])*(b[j]);

        sum+= (a[idx*COLUMNS+0])*(bs[0]);
        sum+= (a[idx*COLUMNS+1])*(bs[1]);
        sum+= (a[idx*COLUMNS+2])*(bs[2]);
        sum+= (a[idx*COLUMNS+3])*(bs[3]);
        sum+= (a[idx*COLUMNS+4])*(bs[4]);
        sum+= (a[idx*COLUMNS+5])*(bs[5]);
        sum+= (a[idx*COLUMNS+6])*(bs[6]);
        sum+= (a[idx*COLUMNS+7])*(bs[7]);
        sum+= (a[idx*COLUMNS+8])*(bs[8]);
        sum+= (a[idx*COLUMNS+9])*(bs[9]);
        sum+= (a[idx*COLUMNS+10])*(bs[10]);
        sum+= (a[idx*COLUMNS+11])*(bs[11]);
        sum+= (a[idx*COLUMNS+12])*(bs[12]);
        sum+= (a[idx*COLUMNS+13])*(bs[13]);
        sum+= (a[idx*COLUMNS+14])*(bs[14]);
        sum+= (a[idx*COLUMNS+15])*(bs[15]);


	c[idx]=sum;
	__syncthreads();

}


int main(int agrc, char * argv[])
{
	unsigned int rows = atoi(argv[1]), i, j; // il y a un malloc contenant ligne et colonnes --> Matrice A et un malloc contenant que colonne -> vecteur B
	int * a_h = (int *) malloc(rows * COLUMNS * sizeof(int)), * b_h = (int *) malloc(COLUMNS * sizeof(int)), * c1_h = (int *) malloc(rows * sizeof(int)), * c2_h = (int *) malloc(rows * sizeof(int));

int *a_d, *b_d, *c_d;

//allocation sur GPU

hipSetDevice (0);

hipMalloc ((void**) &a_d , rows * COLUMNS * sizeof(int));
hipMalloc ((void**) &b_d , COLUMNS * sizeof(int));
hipMalloc ((void**) &c_d , rows * sizeof(int));


//copie vers GPU

hipMemcpy (a_d , a_h , rows * COLUMNS *sizeof(int), hipMemcpyHostToDevice ); // on copie les données du CPU vers le GPU
hipMemcpy (b_d , b_h , COLUMNS * sizeof(int), hipMemcpyHostToDevice );


	unsigned long long ref1, ref2;
	unsigned long long diffH = 0, diffD = 0;
	struct timeval tim;
	
	//remplissage de la matrice

	for(i=0;i<COLUMNS*rows;++i){
		if(i<COLUMNS){
			b_h[i] = i+1;
		}
		a_h[i] = rand()%(COLUMNS*rows);
	}
	
	//multiplication sur CPU

	gettimeofday(&tim, NULL);
	ref1 = tim.tv_sec * 1000000L + tim.tv_usec;
	int * a = a_h, *b, *c=c1_h;
	for(i = 0; i<rows; ++i){
		c1_h[i] = 0;
		for(j = 0; j<COLUMNS;++j,++a,++b)
			c1_h[i] += (a_h[i*COLUMNS+j])*(b_h[j]);
	}
	gettimeofday(&tim, NULL);
	ref2 = tim.tv_sec * 1000000L + tim.tv_usec;
  	diffH+=ref2-ref1; // différence des timing
	
	//multiplication sur GPU

  	gettimeofday(&tim, NULL);
	ref1 = tim.tv_sec * 1000000L + tim.tv_usec;
	
	// EXECUTION GPU  c'est ici que nous allons travailler
	
        int blocks = rows/THREAD_PER_BLOCK;
	multiplication_matrix_GPU<<<blocks,THREAD_PER_BLOCK>>>(a_d, b_d, c_d);
	
	
	
  	gettimeofday(&tim, NULL);
  	ref2 = tim.tv_sec * 1000000L + tim.tv_usec;
  	diffD+=ref2-ref1;

       hipMemcpy(c2_h , c_d , rows * sizeof(int), hipMemcpyDeviceToHost);
  	
	//vérification des résultats et nettoyage
  	int ok = 1;
  	for(i=0;i<10;++i)
  		if(c1_h[i]!=c2_h[i]){
  			//ok = 0;
  			printf("Différence : %d != %d\n", c1_h[i], c2_h[i]);
  		}
  	if(ok)
		printf("Temps de calcul, CPU [%llu usec] GPU [%llu usec] \n", diffH, diffD);
  	
        
	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);

  	free(a_h);
  	free(b_h);
  	free(c1_h);
  	free(c2_h);

}
